#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <ctime>

#define EXP_A (1048576/M_LN2) // 2^20/ln(2) 
#define EXP_C 30801

#define cudaCheck(msg) (cudacheck(msg, __FILE__, __LINE__))

// data buffer and kernel sizes
const size_t N = 8ULL*1024ULL*1024ULL; // 
const size_t MEM_SIZE = N*sizeof(float); // memory required for input vector
const int BLOCK_SIZE = 256;
const int BLOCK_SIZE_2 = 128;
const double VAL = 0.1;

// error handling function
void cudacheck(const char *msg, const char* file, int line) {
        hipError_t __error = hipGetLastError();
        if (__error != hipSuccess) {
                fprintf(stderr, "[FATAL CUDA ERROR] : %s (%s at %s:%d )\n", msg,
                                hipGetErrorString(__error), file, line);
                fprintf(stderr, "***Failed, Aborting***\n");
                exit(1);
        }
}

int ceil_div(int numerator, int denominator) {
        std::div_t res = std::div(numerator, denominator);
        return res.rem? (res.quot + 1) : res.quot;
}

void init_input(double *h_input, int n, double val) {
	for (int i = 0; i < N; i++){
		h_input[i] = ((double) (i - N/2))/(0.1*N); // keep the domain from -5 to 5
	}
}
	
void postprocess(const float ref, const float *res, float ms) {
        bool passed = true;
        if (*res != ref) {
                printf("%25s\n", "*** FAILED ***");
                printf("reference: %f result: %f\n", ref, *res);
                passed = false;
        }
        if (passed == true) {
            printf ("Postprocess passed\n");
            printf("MEMORY SIZE (MBytes): %12.2f, time in ms: %12.4f, \
            Bandwidth (GB/s): %12.4f\n", float(MEM_SIZE)*1e-06, ms, (2*float(MEM_SIZE)*1e-06 )/ ms);
	    printf("------------------------------\n");
        }
}


__device__ inline 
double ecoexp_original_device(double y) {

	if (y > 700 ) return HUGE_VAL;
	if ((y < -700) return 0.0;

	// Original Schraudolph
	union {
		double d;
		struct {
#ifdef LITTLE_ENDIAN
			int j, i;
#else
			int i, j;
#endif
		} n;
	}_eco;

	_eco.n.i = (int) (EXP_A*(y)) + (1072693248 - EXP_C);
	_eco.n.j = 0;

	return _eco.d;
}


__device__ inline 
double ecoexp_schraudolph_corrected_device(double y) {

	if (y > 700 ) return HUGE_VAL;
	if (y < -700) return 0.0;

	// Original Schraudolph
	union {
		double d;
		struct {
#ifdef LITTLE_ENDIAN
			int j, i;
#else
			int i, j;
#endif
		} n;
	}_eco;

	_eco.n.i = (int) (EXP_A*(y)) + (1072693248 - EXP_C);
	_eco.n.j = 0;

	double base =  _eco.d;

	// correction term 
	double y2 = y*y;
	double correction = 1.0 + y2 * (0.0001 + y2 * 0.000001);
	return base*correction;
}

__device__ inline 
double ecoexp_improved1_device(double y) {

	// range reduction
	double k = floor(y / M_LN2 + 0.5); // floor is a CUDA double function
    	double r = y - k * M_LN2;

	// Pade approximation for e^r
	double r2 = r * r;
        double numerator = 2.0 + r + r2 / 6.0;
        double denominator = 2.0 - r + r2 / 6.0;
        double exp_r = numerator / denominator;

	// CUDA's ldexp alternative
	return exp2(k) * exp_r;
}

__device__ inline 
double ecoexp_improved2_device(double y) {

	// range reduction
	double k = floor(y / M_LN2 + 0.5); // floor is a CUDA double function
    	double r = y - k * M_LN2;

	// 5th order polynomial approximation
        double r2 = r * r;
        double r3 = r2 * r;
        double r4 = r2 * r2;
        double r5 = r4 * r;

        double exp_r = 1.0 + r + r2*0.5 + r3/6.0 + r4/24.0 + r5/120.0;

	// CUDA's ldexp alternative
        return exp2(k) * exp_r;
}

// test the kernels
__global__ 

void test_fast_exp_kernels(double* input, double* output_std, 
		double* output_fast, double* errs, int n, int method) {

	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	double result_fast;

	// grid-strided loop
	for (int i = idx; i < n; i += gridDim.x*blockDim.x) {
		double x = input[i];
		double result_std = exp(x)

		// test the various fast kernels
		switch (method) {
			case 0: result_fast = ecoexp_original_device(x); break;
			case 1: result_fast = ecoexp_schraudolph_corrected_device(x); break;
			case 2: result_fast = ecoexp_improved1_device(x); break;
			case 3: result_fast = ecoexp_improved2_device(x); break;
			default: result_fast = ecoexp_improved2_device(x); break;
		}

		output_std[i] = result_std;
		output_fast[i] = result_fast;
		errs[i] = std::abs(result_std - result_fast);

	}

}


// Peeformance kernel
__global__
void bench_fast_exp_kernels(double* input, double* output, 
		int n, int method, int iteration, double* weight) {

	unsogned int idx = lockIdx.x*blovkDim.x + threadIdx.x;
	double result = 0.0f;

	// grid-strided loop
	for (int i = idx; i< n; i+=gridDim.x*blockDim.x) {

		// iterations
		for (int j = 0; j < iterations; j++)n {
			switch (method) {
				case 0: result += exp(x + weight*j); break;  // standard exponential
				case 1: result += ecoexp_improved2_device(x+weight*j); break;  // fast exponential
				default: result += ecoexp_improved2_device(x+weight*j); break;  // fast exponential
			}
		}
		output[i] = result;
	}
}

// Host functions
// Test accuracy of the kernels

void test_accuracy(int method) {

	// buffers
	// host buffers
	double *h_input, *h_output_std, *h_output_fast, *h_errs;
	// device buffers
	double *d_input, *d_output_std, *d_output_fast, *d_errs;
	// allocate host memory
	h_input = new double[N];
	h_output_std = new double[N];
	h_output_fast = new double[N];
	h_errs = new double[N];
	// allocate device memory
	hipMalloc(&d_input, MEM_SIZE);	
	hipMalloc(&d_output_std, MEM_SIZE);	
	hipMalloc(&d_output_fast, MEM_SIZE);
	hipMalloc(&d_errs, MEM_SIZE);
	cudaCheck("cuaMalloc failure");	
	// intialize input data
	init_input(h_input, N, VAL);
	// copy to device
	hipMemcpy(d_input, h_input, MEM_SIZE, cudaMemcpyHosttoDevice);

	// timing variable
	float ms;
	// grid and block dimensions
	dim3 dinGrid(ceil_div(N, BLOCK_SIZE));
	dim3dimBlock(BLOCK_SIZE);
    	printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
    	dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);


